#include "hip/hip_runtime.h"
#include "makeGraph.h"

void makeGraph(int NUM_BLOCKS, int BLOCK_THREADS,float *x, float *y, int numPoints, int minPts, float R, Graph* distGraph, bool** clusterType, report_t *report){

    //Initialize memory for all the arrays
	long unsigned int *dNodes;
	int  *dEdges;
    bool *dClusterType;
    float *d_x, *d_y;

    //hAdjMatrix = (bool*) malloc(sizeof(bool)*(numPoints*numPoints));
    gpuErrchk(hipMalloc((void**)&dNodes, sizeof(long unsigned int) * (numPoints +1)));
    gpuErrchk(hipMalloc((void**)&d_x, sizeof(float) * numPoints));
    gpuErrchk(hipMalloc((void**)&d_y, sizeof(float) * numPoints));
    gpuErrchk(hipMalloc((void**)&dClusterType, sizeof(bool) * numPoints));

    //Copy stuff into the device
    gpuErrchk(hipMemcpy(d_x, x, sizeof(float) * numPoints, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_y, y, sizeof(float) * numPoints, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dClusterType, *clusterType, sizeof(bool) * numPoints, hipMemcpyHostToDevice));

    //Make Adjacency Matrix of all points within radius of all points
    dim3 dimGrid(NUM_BLOCKS,1);
    dim3 dimBlock(BLOCK_THREADS,1);
    fillNodes<<<dimGrid,dimBlock>>>(minPts,R,numPoints,d_x,d_y,dNodes,dClusterType);

    //Get back the info on invididual points (Core or Border)
	gpuErrchk(hipMemcpy(*clusterType, dClusterType, sizeof(bool) * numPoints, hipMemcpyDeviceToHost));
    hipFree(dClusterType);

    //Prints for debugging and monitoring
	if(PRINT_LOG){
		int* hNodes = (int*) malloc(sizeof(int)*(numPoints+1));
		gpuErrchk(hipMemcpy(hNodes, dNodes, sizeof(int) * (numPoints+1), hipMemcpyDeviceToHost));
		printf("Cluster type ");
		for(int i=0;i<numPoints;i++){
			printf("%d ",(*clusterType)[i]);
		}
		printf("\n");
		for(int i=0;i<numPoints;i++){
				   printf("%d " ,hNodes[i]);
		 }
		printf("\nAdj Matrix:");
		free(hNodes);
	}

	//CPU exclusive scan because Thrust was being difficult with large data
//	hNodes[numPoints] = 0;
//	int temp = hNodes[0];
//	int temp2 = hNodes[1];
//	hNodes[0] = 0;
//	hNodes[1] = temp;
//	for(int i=2;i<=numPoints;i++){
//		temp = hNodes[i];
//		hNodes[i] += temp2 + hNodes[i-1];
//		temp2 = temp;
//	}
	//CPU exclusive scan


	//Use Thrust's exclusive scan to make array with neighboring points
    thrust::device_ptr<long unsigned int> in_ptr(dNodes);
    thrust::exclusive_scan(in_ptr, in_ptr + (numPoints + 1), in_ptr);

    //"Get the last element of exclusive scan to allocate memory
    long unsigned int *totalEdges = (long unsigned int*)malloc(sizeof(long unsigned int));
    gpuErrchk(hipMemcpy(totalEdges, &dNodes[numPoints], sizeof(long unsigned int), hipMemcpyDeviceToHost));

    printf("Total edges are %lu\n",(*totalEdges));

    //Save memory by allocating just what is needed
    gpuErrchk(hipMalloc((void**)&dEdges, sizeof(int) * (*totalEdges)));

    //Capture memory info since this is where its consumed most
    size_t free_byte, total_byte ;
	hipMemGetInfo( &free_byte, &total_byte) ;
	set_finalMemory(report,((double)total_byte - (double)free_byte)/1024.0/1024.0);

    //Prints for debugging and monitoring
    if(PRINT_LOG)printf("\nWe have a total of %d edges",*totalEdges);

    //Get adjacency list in contiguous, memory efficient, integer form
    fillEdges<<<dimGrid,dimBlock>>>(numPoints, R,d_x,d_y, dNodes,dEdges);

    //Send the data back
    distGraph->edges = dEdges;
    distGraph->nodes = dNodes;
    distGraph->totalEdges = *totalEdges;

    //Clean up the mess
    hipFree(d_x);
    hipFree(d_y);

}

__global__ void fillNodes(int minPts, float R, int numPoints, float *d_x,
								float *d_y,long unsigned int * dNodes, bool* dClusterType){
    //Grid Stride Loop
    for (int tID = blockIdx.x * blockDim.x + threadIdx.x; 
         tID < numPoints;
         tID += blockDim.x * gridDim.x){

        float thisPointX = d_x[tID];
        float thisPointY = d_y[tID];
        float distance;
        dNodes[tID] = 0;
        __syncthreads();
        for(int i=0; i<numPoints; i++){
            distance = euclidean_distance(thisPointX,thisPointY,d_x[i],d_y[i]);
            if(distance <= R){
                dNodes[tID]++;
            }
        }
        __syncthreads();

        if(dNodes[tID]>=minPts){
        	dClusterType[tID] = Core;
        }
    }
}


__global__ void fillEdges(int numPoints, float R,float *d_x,float *d_y, long unsigned int * dNodes,int *dEdges){
	for (int tID = blockIdx.x * blockDim.x + threadIdx.x;
		          tID < numPoints;
		          tID += blockDim.x * gridDim.x)
	{
        float thisPointX = d_x[tID];
        float thisPointY = d_y[tID];
        float distance;
		int edgeOffset = dNodes[tID];
		__syncthreads();
		for(int i=0; i<numPoints; i++){
			distance = euclidean_distance(thisPointX,thisPointY,d_x[i],d_y[i]);
			if(distance <= R){
				dEdges[edgeOffset] = i;
				edgeOffset++;
			}
		}
	}

}
__device__ __host__
float euclidean_distance(float p1_x, float p1_y,
                         float p2_x, float p2_y)
{
    float x_diff = p1_x - p2_x;
    float y_diff = p1_y - p2_y;
    return sqrt(x_diff * x_diff + y_diff * y_diff);
}
