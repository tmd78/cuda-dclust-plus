#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <stdio.h>
#include <stdlib.h>
#include <thrust/binary_search.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/sort.h>
#include <time.h>

#include <algorithm>
#include <ctime>
#include <fstream>
#include <map>
#include <math.h>
#include <set>
#include <vector>

using namespace std;

#define RANGE 2
#define UNPROCESSED -1
#define NOISE -2

#define DIMENSION 2
#define TREE_LEVELS (DIMENSION + 1)

#define THREAD_BLOCKS 256
#define THREAD_COUNT 256

#define MAX_SEEDS 1024

__managed__ int MINPTS = 4;
__managed__ double EPS = 1.5;
__managed__ int DATASET_COUNT = 400000;
__managed__ int PARTITION_SIZE = 80;

#define POINTS_SEARCHED 9

#define PORTO 1
#define SPATIAL 0
#define NGSI 0
#define IONO2D 0
#define SPATIAL3D 0
#define IONO3D 0

struct __align__(8) IndexStructure {
  int dimension;
  int dataBegin;
  int dataEnd;
  int childFrom;
};

/**
**************************************************************************
//////////////////////////////////////////////////////////////////////////
* GPU ERROR function checks for potential erros in cuda function execution
//////////////////////////////////////////////////////////////////////////
**************************************************************************
*/
#define gpuErrchk(ans) \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort) exit(code);
  }
}

/**
**************************************************************************
//////////////////////////////////////////////////////////////////////////
* Declare CPU and GPU Functions
//////////////////////////////////////////////////////////////////////////
**************************************************************************
*/
int ImportDataset(char const *fname, double *dataset);

bool MonitorSeedPoints(vector<int>& unprocessedPoints, int* runningCluster,
                       int* d_cluster, int* d_seedList, int* d_seedLength,
                       int* d_collisionMatrix, int* d_results);

void GetDbscanResult(int* d_cluster, int* runningCluster, int* clusterCount,
                     int* noiseCount);

__global__ void DBSCAN(double* dataset, int* cluster, int* seedList,
                       int* seedLength, int* collisionMatrix,
                       int* results,
                       struct IndexStructure** indexBuckets,
                       int* indexesStack, int* dataValue, double* upperBounds,
                       double* binWidth, double *minPoints, double *maxPoints);

__device__ void MarkAsCandidate(int neighborID, int chainID, int* cluster,
                                int* seedList, int* seedLength, int* collisionMatrix);

                                __global__ void INDEXING_ADJUSTMENT(int *indexTreeMetaData,
                                    struct IndexStructure **indexBuckets,
                                    int *dataKey);

__global__ void INDEXING_STRUCTURE(double *dataset, int *indexTreeMetaData,
                                        double *minPoints, double *maxPoints,
                                        double *binWidth, int *results,
                                        struct IndexStructure **indexBuckets,
                                        int *dataKey, int *dataValue,
                                        double *upperBounds);

__device__ void insertData(int id, double *dataset,
                                struct IndexStructure **indexBuckets,
                                int *dataKey, int *dataValue,
                                double *upperBounds, double *binWidth,
                                double *minPoints, double *maxPoints);

__device__ void indexConstruction(int level, int *indexTreeMetaData,
                                       double *minPoints, double *binWidth,
                                       struct IndexStructure **indexBuckets,
                                       double *upperBounds);

__device__ void searchPoints(double *data, int chainID, double *dataset,
                                  int *results,
                                  struct IndexStructure **indexBuckets,
                                  int *indexesStack, int *dataValue,
                                  double *upperBounds, double *binWidth,
                                  double *minPoints, double *maxPoints);



void runDBSCAN(const char* filename, int datasetSize, double eps, int minPts, int partition) {

  EPS = eps;
  MINPTS = minPts;
  DATASET_COUNT = datasetSize;
  PARTITION_SIZE = partition;


  printf("Using dataset file %s\n", filename);
  double *importedDataset =
      (double *)malloc(sizeof(double) * DATASET_COUNT * DIMENSION);

  // Import data from dataset
  int ret = ImportDataset(filename, importedDataset);
  if (ret == 1) {
    printf("\nError importing the dataset");
    return;
  }

  // Get the total count of dataset
  vector<int> unprocessedPoints;
  for (int x = 0; x < DATASET_COUNT; x++) {
    unprocessedPoints.push_back(x);
  }

  // Reset the GPU device for potential memory issues
  gpuErrchk(hipDeviceReset());
  gpuErrchk(hipFree(0));

  // Start the time
  clock_t totalTimeStart, totalTimeStop, indexingStart, indexingStop;
  float totalTime = 0.0;
  float indexingTime = 0.0;
  totalTimeStart = clock();

  /**
 **************************************************************************
 * CUDA Memory allocation
 **************************************************************************
 */

  double *d_dataset;
  int *d_cluster;
  int *d_seedList;
  int *d_seedLength;
  int *d_collisionMatrix;

  gpuErrchk(hipMalloc((void **)&d_dataset,
                       sizeof(double) * DATASET_COUNT * DIMENSION));

  gpuErrchk(hipMalloc((void **)&d_cluster, sizeof(int) * DATASET_COUNT));

  gpuErrchk(hipMalloc((void **)&d_seedList,
                       sizeof(int) * THREAD_BLOCKS * MAX_SEEDS));

  gpuErrchk(hipMalloc((void **)&d_seedLength, sizeof(int) * THREAD_BLOCKS));

  gpuErrchk(hipMalloc((void **)&d_collisionMatrix,
                       sizeof(int) * THREAD_BLOCKS * THREAD_BLOCKS));


  /**
 **************************************************************************
 * Indexing Memory allocation
 **************************************************************************
 */

  indexingStart = clock();

  int *d_indexTreeMetaData;
  int *d_results;
  double *d_minPoints;
  double *d_maxPoints;
  double *d_binWidth;

  gpuErrchk(hipMalloc((void **)&d_indexTreeMetaData,
                       sizeof(int) * TREE_LEVELS * RANGE));

  gpuErrchk(hipMalloc((void **)&d_results,
                       sizeof(int) * THREAD_BLOCKS * POINTS_SEARCHED));

  gpuErrchk(hipMalloc((void **)&d_minPoints, sizeof(double) * DIMENSION));
  gpuErrchk(hipMalloc((void **)&d_maxPoints, sizeof(double) * DIMENSION));

  gpuErrchk(hipMalloc((void **)&d_binWidth, sizeof(double) * DIMENSION));

  gpuErrchk(
      hipMemset(d_results, -1, sizeof(int) * THREAD_BLOCKS * POINTS_SEARCHED));

  /**
 **************************************************************************
 * Assignment with default values
 **************************************************************************
 */
  gpuErrchk(hipMemcpy(d_dataset, importedDataset,
                       sizeof(double) * DATASET_COUNT * DIMENSION,
                       hipMemcpyHostToDevice));

  gpuErrchk(hipMemset(d_cluster, UNPROCESSED, sizeof(int) * DATASET_COUNT));

  gpuErrchk(
      hipMemset(d_seedList, -1, sizeof(int) * THREAD_BLOCKS * MAX_SEEDS));

  gpuErrchk(hipMemset(d_seedLength, 0, sizeof(int) * THREAD_BLOCKS));

  gpuErrchk(hipMemset(d_collisionMatrix, -1,
                       sizeof(int) * THREAD_BLOCKS * THREAD_BLOCKS));

  /**
**************************************************************************
* Initialize index structure
**************************************************************************
*/
  double maxPoints[DIMENSION];
  double minPoints[DIMENSION];

  for (int j = 0; j < DIMENSION; j++) {
    maxPoints[j] = 0;
    minPoints[j] = 999999999;
  }

  for (int i = 0; i < DATASET_COUNT; i++) {
    for (int j = 0; j < DIMENSION; j++) {
      if (importedDataset[i * DIMENSION + j] > maxPoints[j]) {
        maxPoints[j] = importedDataset[i * DIMENSION + j];
      }
      if (importedDataset[i * DIMENSION + j] < minPoints[j]) {
        minPoints[j] = importedDataset[i * DIMENSION + j];
      }
    }
  }

  double binWidth[DIMENSION];
  double minBinSize = 99999999;
  for (int x = 0; x < DIMENSION; x++) {
    binWidth[x] = (double)(maxPoints[x] - minPoints[x]) / PARTITION_SIZE;
    if (minBinSize >= binWidth[x]) {
      minBinSize = binWidth[x];
    }
  }

  if (minBinSize < EPS) {
    printf("Bin width (%f) is less than EPS(%f).\n", minBinSize, EPS);
    exit(0);
  }

  // Level Partition
  int treeLevelPartition[TREE_LEVELS] = {1};

  for (int i = 0; i < DIMENSION; i++) {
    treeLevelPartition[i + 1] = PARTITION_SIZE;
  }

  int childItems[TREE_LEVELS];
  int startEndIndexes[TREE_LEVELS * RANGE];

  int mulx = 1;
  for (int k = 0; k < TREE_LEVELS; k++) {
    mulx *= treeLevelPartition[k];
    childItems[k] = mulx;
  }

  for (int i = 0; i < TREE_LEVELS; i++) {
    if (i == 0) {
      startEndIndexes[i * RANGE + 0] = 0;
      startEndIndexes[i * RANGE + 1] = 1;
      continue;
    }
    startEndIndexes[i * RANGE + 0] = startEndIndexes[((i - 1) * RANGE) + 1];
    startEndIndexes[i * RANGE + 1] = startEndIndexes[i * RANGE + 0];
    for (int k = 0; k < childItems[i - 1]; k++) {
      startEndIndexes[i * RANGE + 1] += treeLevelPartition[i];
    }
  }

  gpuErrchk(hipMemcpy(d_minPoints, minPoints, sizeof(double) * DIMENSION,
                       hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_maxPoints, maxPoints, sizeof(double) * DIMENSION,
  hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_binWidth, binWidth, sizeof(double) * DIMENSION,
                       hipMemcpyHostToDevice));

  gpuErrchk(hipMemcpy(d_indexTreeMetaData, startEndIndexes,
                       sizeof(int) * TREE_LEVELS * RANGE,
                       hipMemcpyHostToDevice));

  int indexedStructureSize = startEndIndexes[DIMENSION * RANGE + 1];

  // Allocate memory for index buckets
  struct IndexStructure **d_indexBuckets, *d_currentIndexBucket;

  gpuErrchk(hipMalloc((void **)&d_indexBuckets,
                       sizeof(struct IndexStructure *) * indexedStructureSize));

  for (int i = 0; i < indexedStructureSize; i++) {
    gpuErrchk(hipMalloc((void **)&d_currentIndexBucket,
                         sizeof(struct IndexStructure)));
    gpuErrchk(hipMemcpy(&d_indexBuckets[i], &d_currentIndexBucket,
                         sizeof(struct IndexStructure *),
                         hipMemcpyHostToDevice));
  }

  // Allocate memory for current indexes stack
  int indexBucketSize = 1;
  for (int i = 0; i < DIMENSION; i++) {
    indexBucketSize *= 3;
  }

  indexBucketSize = indexBucketSize * THREAD_BLOCKS;

  int *d_indexesStack;

  gpuErrchk(
      hipMalloc((void **)&d_indexesStack, sizeof(int) * indexBucketSize));

  hipFree(d_currentIndexBucket);

  /**
 **************************************************************************
 * Data key-value pair
 **************************************************************************
 */
  int *d_dataKey;
  int *d_dataValue;
  double *d_upperBounds;

  gpuErrchk(hipMalloc((void **)&d_dataKey, sizeof(int) * DATASET_COUNT));
  gpuErrchk(hipMalloc((void **)&d_dataValue, sizeof(int) * DATASET_COUNT));
  gpuErrchk(hipMalloc((void **)&d_upperBounds,
                       sizeof(double) * indexedStructureSize));
  /**
 **************************************************************************
 * Start Indexing first
 **************************************************************************
 */

 
  hipDeviceSetLimit(hipLimitMallocHeapSize, 16*1024*1024);
  gpuErrchk(hipDeviceSynchronize());

  INDEXING_STRUCTURE<<<dim3(THREAD_BLOCKS, 1), dim3(THREAD_COUNT, 1)>>>(
      d_dataset, d_indexTreeMetaData, d_minPoints, d_maxPoints, d_binWidth, d_results,
      d_indexBuckets, d_dataKey, d_dataValue, d_upperBounds);
  gpuErrchk(hipDeviceSynchronize());

  hipFree(d_indexTreeMetaData);
  

  /**
 **************************************************************************
 * Sorting and adjusting Data key-value pair
 **************************************************************************
 */

  thrust::sort_by_key(thrust::device, d_dataKey, d_dataKey + DATASET_COUNT,
                      d_dataValue);

  gpuErrchk(hipDeviceSynchronize());

  INDEXING_ADJUSTMENT<<<dim3(THREAD_BLOCKS, 1), dim3(THREAD_COUNT, 1)>>>(
      d_indexTreeMetaData, d_indexBuckets, d_dataKey);

  gpuErrchk(hipDeviceSynchronize());

  indexingStop = clock();

  /**
 **************************************************************************
 * Start the DBSCAN algorithm
 **************************************************************************
 */

  // Keep track of number of cluster formed without global merge
  int runningCluster = THREAD_BLOCKS;
  // Global cluster count
  int clusterCount = 0;

  // Keeps track of number of noises
  int noiseCount = 0;

  // Handler to conmtrol the while loop
  bool exit = false;

  clock_t communicationStart, communicationStop, dbscanKernelStart, dbscanKernelStop;
  float communicationTime = 0.0;
  float dbscanKernelTime = 0.0;

  while (!exit) {

    communicationStart = clock();
    // Monitor the seed list and return the comptetion status of points
    int completed =
        MonitorSeedPoints(unprocessedPoints, &runningCluster,
                          d_cluster, d_seedList, d_seedLength,
                          d_collisionMatrix, d_results);

    communicationStop = clock();
    communicationTime += (float)(communicationStop - communicationStart) / CLOCKS_PER_SEC;

    // If all points are processed, exit
    if (completed) {
      exit = true;
    }

    if (exit) break;

    dbscanKernelStart = clock();

    // Kernel function to expand the seed list
    gpuErrchk(hipDeviceSynchronize());
    DBSCAN<<<dim3(THREAD_BLOCKS, 1), dim3(THREAD_COUNT, 1)>>>(
        d_dataset, d_cluster, d_seedList, d_seedLength, d_collisionMatrix,
        d_results, d_indexBuckets, d_indexesStack,
        d_dataValue, d_upperBounds, d_binWidth, d_minPoints, d_maxPoints);
    gpuErrchk(hipDeviceSynchronize());

    dbscanKernelStop = clock();
    dbscanKernelTime += (float)(dbscanKernelStop - dbscanKernelStart) / CLOCKS_PER_SEC;
  }

  /**
 **************************************************************************
 * End DBSCAN and show the results
 **************************************************************************
 */
  totalTimeStop = clock();
  
  // Get the DBSCAN result
  GetDbscanResult(d_cluster, &runningCluster, &clusterCount, &noiseCount);
  
  totalTime = (float)(totalTimeStop - totalTimeStart) / CLOCKS_PER_SEC;
  indexingTime = (float)(indexingStop - indexingStart) / CLOCKS_PER_SEC;

  printf("==============================================\n");
  printf("Dataset: %d\nEPS: %f\nMINPTS: %d\nPARTITION: %d\n", DATASET_COUNT, EPS, MINPTS, PARTITION_SIZE);
  printf("Final cluster after merging: %d\n", clusterCount);
  printf("Number of noises: %d\n", noiseCount);
  printf("Indexing Time: %3.2f seconds\n", indexingTime);
  printf("Communication Time: %3.2f seconds\n", communicationTime);
  printf("DBSCAN kernel Time: %3.2f seconds\n", dbscanKernelTime);
  printf("Total Time: %3.2f seconds\n", totalTime);
  printf("==============================================\n");

  /**
 **************************************************************************
 * Free CUDA memory allocations
 **************************************************************************
 */

  hipFree(d_dataset);
  hipFree(d_cluster);
  hipFree(d_seedList);
  hipFree(d_seedLength);
  hipFree(d_collisionMatrix);

  hipFree(d_results);
  hipFree(d_indexBuckets);
  hipFree(d_indexesStack);

  hipFree(d_dataKey);
  hipFree(d_dataValue);
  hipFree(d_upperBounds);
  hipFree(d_binWidth);

  hipFree(d_minPoints);
  hipFree(d_maxPoints);
}


__global__ void DBSCAN(double *dataset, int *cluster, int *seedList,
                       int *seedLength, int *collisionMatrix,
                      int *results,
                       struct IndexStructure **indexBuckets,
                       int *indexesStack, int *dataValue, double *upperBounds,
                       double *binWidth, double *minPoints, double *maxPoints) {
  // Point ID to expand by a block
  __shared__ int pointID;

  // Neighbors to store of neighbors points exceeds minpoints
  __shared__ int neighborBuffer[64];

  // It counts the total neighbors
  __shared__ int neighborCount;

  // ChainID is basically blockID
  __shared__ int chainID;

  // Store the point from pointID
  __shared__ double point[DIMENSION];

  // Length of the seedlist to check its size
  __shared__ int currentSeedLength;

  __shared__ int resultId;

  if (threadIdx.x == 0) {
    chainID = blockIdx.x;
    currentSeedLength = seedLength[chainID];
    pointID = seedList[chainID * MAX_SEEDS + currentSeedLength - 1];
  }
  __syncthreads();


    // Check if the point is already processed
    if (threadIdx.x == 0) {
      seedLength[chainID] = currentSeedLength - 1;
      neighborCount = 0;
      for (int x = 0; x < DIMENSION; x++) {
        point[x] = dataset[pointID * DIMENSION + x];
      }
    }
    __syncthreads();

    ///////////////////////////////////////////////////////////////////////////////////

    searchPoints(point, chainID, dataset, results, indexBuckets, indexesStack,
                 dataValue, upperBounds, binWidth, minPoints, maxPoints);

    __syncthreads();

    for (int k = 0; k < POINTS_SEARCHED; k++) {
      if (threadIdx.x == 0) {
        resultId = results[chainID * POINTS_SEARCHED + k];
      }
      __syncthreads();

      if (resultId == -1) break;

      for (int i = threadIdx.x + indexBuckets[resultId]->dataBegin;
           i < indexBuckets[resultId]->dataEnd; i = i + THREAD_COUNT) {
        register double comparingPoint[DIMENSION];

        for (int x = 0; x < DIMENSION; x++) {
          comparingPoint[x] = dataset[dataValue[i] * DIMENSION + x];
        }

        register double distance = 0;
        for (int x = 0; x < DIMENSION; x++) {
          distance +=
              (point[x] - comparingPoint[x]) * (point[x] - comparingPoint[x]);
        }

        if (distance <= EPS * EPS) {
          register int currentNeighborCount = atomicAdd(&neighborCount, 1);
          if (currentNeighborCount >= MINPTS) {
            MarkAsCandidate(dataValue[i], chainID, cluster, seedList,
                            seedLength, collisionMatrix);
          } else {
            neighborBuffer[currentNeighborCount] = dataValue[i];
          }
        }
      }
      __syncthreads();
    }
    __syncthreads();

    ///////////////////////////////////////////////////////////////////////////////////

    if (neighborCount >= MINPTS) {
      cluster[pointID] = chainID;
      for (int i = threadIdx.x; i < MINPTS; i = i + THREAD_COUNT) {
        MarkAsCandidate(neighborBuffer[i], chainID, cluster, seedList,
                        seedLength, collisionMatrix);
      }
    } else {
      cluster[pointID] = NOISE;
    }

    __syncthreads();
    ///////////////////////////////////////////////////////////////////////////////////

    if (threadIdx.x == 0 && seedLength[chainID] >= MAX_SEEDS) {
      seedLength[chainID] = MAX_SEEDS - 1;
    }
    __syncthreads();
  
}

bool MonitorSeedPoints(vector<int> &unprocessedPoints, int *runningCluster,
                       int *d_cluster, int *d_seedList, int *d_seedLength,
                       int *d_collisionMatrix,
                       int *d_results) {
  int *localSeedLength;
  localSeedLength = (int *)malloc(sizeof(int) * THREAD_BLOCKS);
  gpuErrchk(hipMemcpy(localSeedLength, d_seedLength,
                       sizeof(int) * THREAD_BLOCKS, hipMemcpyDeviceToHost));

  int *localSeedList;
  localSeedList = (int *)malloc(sizeof(int) * THREAD_BLOCKS * MAX_SEEDS);
  gpuErrchk(hipMemcpy(localSeedList, d_seedList,
                       sizeof(int) * THREAD_BLOCKS * MAX_SEEDS,
                       hipMemcpyDeviceToHost));

  int *localCollisionMatrix;
  localCollisionMatrix =
      (int *)malloc(sizeof(int) * THREAD_BLOCKS * THREAD_BLOCKS);
  gpuErrchk(hipMemcpy(localCollisionMatrix, d_collisionMatrix,
                       sizeof(int) * THREAD_BLOCKS * THREAD_BLOCKS,
                       hipMemcpyDeviceToHost));

  gpuErrchk(
      hipMemset(d_results, -1, sizeof(int) * THREAD_BLOCKS * POINTS_SEARCHED));

  int completeSeedListFirst = false;
  for (int i = 0; i < THREAD_BLOCKS; i++) {
    if (localSeedLength[i] > 0) {
      completeSeedListFirst = true;
    }
  }
  if (completeSeedListFirst) {
    free(localSeedList);
    free(localSeedLength);
    return false;
  }
  ////////////////////////////////////////////////////////////////////////////////////////

  int clusterMap[THREAD_BLOCKS];
  set<int> blockSet;
  for (int i = 0; i < THREAD_BLOCKS; i++) {
    blockSet.insert(i);
  }

  set<int>::iterator it;

  while (blockSet.empty() == 0) {
    it = blockSet.begin();
    int curBlock = *it;
    set<int> expansionQueue;
    set<int> finalQueue;

    expansionQueue.insert(curBlock);
    finalQueue.insert(curBlock);

    while (expansionQueue.empty() == 0) {
      it = expansionQueue.begin();
      int expandBlock = *it;
      expansionQueue.erase(it);
      blockSet.erase(expandBlock);
      for (int x = 0; x < THREAD_BLOCKS; x++) {
        if (x == expandBlock) continue;
        if (localCollisionMatrix[expandBlock * THREAD_BLOCKS + x] == 1 &&
            blockSet.find(x) != blockSet.end()) {
          expansionQueue.insert(x);
          finalQueue.insert(x);
        }
      }
    }

    for (it = finalQueue.begin(); it != finalQueue.end(); ++it) {
      clusterMap[*it] = curBlock;
    }
  }

  int clusterCountMap[THREAD_BLOCKS];
  for (int x = 0; x < THREAD_BLOCKS; x++) {
    clusterCountMap[x] = UNPROCESSED;
  }

  for (int x = 0; x < THREAD_BLOCKS; x++) {
    if (clusterCountMap[clusterMap[x]] != UNPROCESSED) continue;
    clusterCountMap[clusterMap[x]] = (*runningCluster);
    (*runningCluster)++;
  }

  for (int x = 0; x < THREAD_BLOCKS; x++) {
    thrust::replace(thrust::device, d_cluster, d_cluster + DATASET_COUNT, x,
                    clusterCountMap[clusterMap[x]]);
  }

  //////////////////////////////////////////////////////////////////////////////////////////

  int *localCluster;
  localCluster = (int *)malloc(sizeof(int) * DATASET_COUNT);
  gpuErrchk(hipMemcpy(localCluster, d_cluster, sizeof(int) * DATASET_COUNT,
                       hipMemcpyDeviceToHost));

  int complete = 0;
  for (int i = 0; i < THREAD_BLOCKS; i++) {
    bool found = false;
    while (!unprocessedPoints.empty()) {
      int lastPoint = unprocessedPoints.back();
      unprocessedPoints.pop_back();

      if (localCluster[lastPoint] == UNPROCESSED) {
        localSeedLength[i] = 1;
        localSeedList[i * MAX_SEEDS] = lastPoint;
        found = true;
        break;
      }
    }

    if (!found) {
      complete++;
    }
  }

  // FInally, transfer back the CPU memory to GPU and run DBSCAN process

  gpuErrchk(hipMemcpy(d_seedLength, localSeedLength,
                       sizeof(int) * THREAD_BLOCKS, hipMemcpyHostToDevice));

  gpuErrchk(hipMemcpy(d_seedList, localSeedList,
                       sizeof(int) * THREAD_BLOCKS * MAX_SEEDS,
                       hipMemcpyHostToDevice));

  gpuErrchk(hipMemset(d_collisionMatrix, -1,
                       sizeof(int) * THREAD_BLOCKS * THREAD_BLOCKS));

  // Free CPU memories

  free(localCluster);
  free(localSeedList);
  free(localSeedLength);
  free(localCollisionMatrix);

  if (complete == THREAD_BLOCKS) {
    return true;
  }

  return false;
}

__device__ void MarkAsCandidate(int neighborID, int chainID, int *cluster,
                                int *seedList, int *seedLength,
                                int *collisionMatrix) {
  register int oldState =
      atomicCAS(&(cluster[neighborID]), UNPROCESSED, chainID);

  if (oldState == UNPROCESSED) {
    register int sl = atomicAdd(&(seedLength[chainID]), 1);
    if (sl < MAX_SEEDS) {
      seedList[chainID * MAX_SEEDS + sl] = neighborID;
    }
  }
 else if (oldState != NOISE && oldState != chainID &&
           oldState < THREAD_BLOCKS) {
    collisionMatrix[oldState * THREAD_BLOCKS + chainID] = 1;
    collisionMatrix[chainID * THREAD_BLOCKS + oldState] = 1;
  }

  else if (oldState == NOISE) {
    oldState = atomicCAS(&(cluster[neighborID]), NOISE, chainID);
  }
}

void GetDbscanResult(int *d_cluster, int *runningCluster, int *clusterCount,
                     int *noiseCount) {
  *noiseCount = thrust::count(thrust::device, d_cluster, d_cluster + DATASET_COUNT, NOISE);
  int *d_localCluster;
  gpuErrchk(hipMalloc((void **)&d_localCluster, sizeof(int) * DATASET_COUNT));
  thrust::copy(thrust::device, d_cluster, d_cluster + DATASET_COUNT, d_localCluster);
  thrust::sort(thrust::device, d_localCluster, d_localCluster + DATASET_COUNT);
  *clusterCount = thrust::unique(thrust::device, d_localCluster, d_localCluster + DATASET_COUNT) - d_localCluster - 1;
  


  int *localCluster;
  localCluster = (int *)malloc(sizeof(int) * DATASET_COUNT);
  gpuErrchk(hipMemcpy(localCluster, d_localCluster, sizeof(int) * DATASET_COUNT,
                        hipMemcpyDeviceToHost));
  ofstream outputFile;
  outputFile.open("./out/cuda_dclust_extended.txt");
  for (int j = 0; j < DATASET_COUNT; j++) {
    outputFile << localCluster[j] << endl;
  }
  outputFile.close();
  free(localCluster);


  hipFree(d_localCluster);
}

__device__ void searchPoints(double *data, int chainID, double *dataset,
                                  int *results,
                                  struct IndexStructure **indexBuckets,
                                  int *indexesStack, int *dataValue,
                                  double *upperBounds, double *binWidth, double *minPoints, double *maxPoints) {

  __shared__ int resultsCount;
  __shared__ int indexBucketSize;
  __shared__ int currentIndex;
  __shared__ int currentIndexSize;
  __shared__ double comparingData;

  if (threadIdx.x == 0) {
    resultsCount = 0;
    indexBucketSize = 1;
    for (int i = 0; i < DIMENSION; i++) {
      indexBucketSize *= 3;
    }
    indexBucketSize = indexBucketSize * chainID;
    currentIndexSize = indexBucketSize;
    indexesStack[currentIndexSize++] = 0;
  }
  __syncthreads();

  while (currentIndexSize > indexBucketSize) {
    if (threadIdx.x == 0) {
      currentIndexSize = currentIndexSize - 1;
      currentIndex = indexesStack[currentIndexSize];
      comparingData = data[indexBuckets[currentIndex]->dimension];
    }
    __syncthreads();

    for (int k = threadIdx.x + indexBuckets[currentIndex]->childFrom;
         k < indexBuckets[currentIndex]->childFrom + PARTITION_SIZE;
         k = k + THREAD_COUNT) {
      double leftRange;
      double rightRange;
      if (k == indexBuckets[currentIndex]->childFrom) {
        leftRange =
            upperBounds[k] - binWidth[indexBuckets[currentIndex]->dimension];
      } else {
        leftRange = upperBounds[k - 1];
      }

      rightRange = upperBounds[k];

      if (comparingData >= leftRange && comparingData < rightRange) {
        if (indexBuckets[currentIndex]->dimension == DIMENSION - 1) {
          int oldResultsCount = atomicAdd(&resultsCount, 1);
          results[chainID * POINTS_SEARCHED + oldResultsCount] = k;

          if (k > indexBuckets[currentIndex]->childFrom) {
            oldResultsCount = atomicAdd(&resultsCount, 1);
            results[chainID * POINTS_SEARCHED + oldResultsCount] = k - 1;
          }

          if (k < indexBuckets[currentIndex]->childFrom + PARTITION_SIZE - 1) {
            oldResultsCount = atomicAdd(&resultsCount, 1);
            results[chainID * POINTS_SEARCHED + oldResultsCount] = k + 1;
          }
        } else {
          int oldCurrentIndexSize = atomicAdd(&currentIndexSize, 1);
          indexesStack[oldCurrentIndexSize] = k;
          if (k > indexBuckets[currentIndex]->childFrom) {
            int oldCurrentIndexSize = atomicAdd(&currentIndexSize, 1);
            indexesStack[oldCurrentIndexSize] = k - 1;
          }
          if (k < indexBuckets[currentIndex]->childFrom + PARTITION_SIZE - 1) {
            int oldCurrentIndexSize = atomicAdd(&currentIndexSize, 1);
            indexesStack[oldCurrentIndexSize] = k + 1;
          }
        }
      }
    }

    __syncthreads();
  }
}



__device__ void indexConstruction(int level, int *indexTreeMetaData,
                                  double *minPoints, double *binWidth,
                                  struct IndexStructure **indexBuckets,
                                  double *upperBounds) {
  for (int k = threadIdx.x + indexTreeMetaData[level * RANGE + 0];
       k < indexTreeMetaData[level * RANGE + 1]; k = k + THREAD_COUNT) {
    for (int i = 0; i < PARTITION_SIZE; i++) {
      int currentBucketIndex =
          indexTreeMetaData[level * RANGE + 1] + i +
          (k - indexTreeMetaData[level * RANGE + 0]) * PARTITION_SIZE;

      indexBuckets[k]->dimension = level;
      indexBuckets[currentBucketIndex]->dimension = level + 1;

      if (i == 0) {
        indexBuckets[k]->childFrom = currentBucketIndex;
      }

      double rightPoint =
          minPoints[level] + i * binWidth[level] + binWidth[level];

      if (i == PARTITION_SIZE - 1) rightPoint = rightPoint + binWidth[level];

      upperBounds[currentBucketIndex] = rightPoint;
    }
  }
  __syncthreads();
}

__device__ void insertData(int id, double *dataset,
                           struct IndexStructure **indexBuckets, int *dataKey,
                           int *dataValue, double *upperBounds,
                           double *binWidth, double *minPoints, double *maxPoints) {
  int index = 0;
  for (int j = 0; j < DIMENSION; j++) {
    double x = dataset[id * DIMENSION + j];
    int currentIndex = (x - minPoints[j]) / (maxPoints[j] - minPoints[j]) * PARTITION_SIZE + 1;
    index = index * PARTITION_SIZE + currentIndex;
  }

  dataValue[id] = id;
  dataKey[id] = index;
}


__global__ void INDEXING_ADJUSTMENT(int *indexTreeMetaData,
                                    struct IndexStructure **indexBuckets,
                                    int *dataKey) {
  __shared__ int indexingRange;
  if (threadIdx.x == 0) {
    indexingRange = indexTreeMetaData[DIMENSION * RANGE + 1] -
                    indexTreeMetaData[DIMENSION * RANGE];
  }
  __syncthreads();

  int threadId = blockDim.x * blockIdx.x + threadIdx.x;

  for (int i = threadId; i < indexingRange;
       i = i + THREAD_COUNT * THREAD_BLOCKS) {
    int idx = indexTreeMetaData[DIMENSION * RANGE] + i;

    thrust::pair<int *, int *> dataPositioned;

    dataPositioned = thrust::equal_range(thrust::device, dataKey, dataKey + DATASET_COUNT, idx);

    indexBuckets[idx]->dataBegin = dataPositioned.first - dataKey;
    indexBuckets[idx]->dataEnd = dataPositioned.second - dataKey;
  }
  __syncthreads();
}


__global__ void INDEXING_STRUCTURE(double *dataset, int *indexTreeMetaData,
                                   double *minPoints, double *maxPoints, double *binWidth,
                                   int *results,
                                   struct IndexStructure **indexBuckets,
                                   int *dataKey, int *dataValue,
                                   double *upperBounds) {
  if (blockIdx.x < DIMENSION) {
    indexConstruction(blockIdx.x, indexTreeMetaData, minPoints, binWidth,
                      indexBuckets, upperBounds);
  }
  __syncthreads();

  int threadId = blockDim.x * blockIdx.x + threadIdx.x;
  for (int i = threadId; i < DATASET_COUNT;
       i = i + THREAD_COUNT * THREAD_BLOCKS) {
    insertData(i, dataset, indexBuckets, dataKey, dataValue, upperBounds,
               binWidth, minPoints, maxPoints);
  }
  __syncthreads();
}


/**
**************************************************************************
//////////////////////////////////////////////////////////////////////////
* Import Dataset
* It imports the data from the file and store in dataset variable
//////////////////////////////////////////////////////////////////////////
**************************************************************************
*/
int ImportDataset(char const *fname, double *dataset) {
  FILE *fp = fopen(fname, "r");
  if (!fp) {
    printf("Unable to open file\n");
    return (1);
  }

  char buf[4096];
  unsigned long int cnt = 0;
  while (fgets(buf, 4096, fp) && cnt < DATASET_COUNT * DIMENSION) {
    char *field = strtok(buf, ",");
    long double tmp;
    sscanf(field, "%Lf", &tmp);
    dataset[cnt] = tmp;
    cnt++;

    while (field) {
      field = strtok(NULL, ",");

      if (field != NULL) {
        long double tmp;
        sscanf(field, "%Lf", &tmp);
        dataset[cnt] = tmp;
        cnt++;
      }
    }
  }
  fclose(fp);
  return 0;
}


/**
**************************************************************************
//////////////////////////////////////////////////////////////////////////
* Main CPU function
//////////////////////////////////////////////////////////////////////////
**************************************************************************
*/

int main() {

 // Generate random datasets
 char *datasetPath;
  double setOfR[5];
  int setOfMinPts[5];
  int defaultMin, defaultPts;
  double defaultR;
  int defaultP;
  int setOfDataSize[5];
  int setOfP[5];

  if (PORTO) {
    setOfDataSize[0] = 40000;
    setOfDataSize[1] = 80000;
    setOfDataSize[2] = 160000;
    setOfDataSize[3] = 320000;
    setOfDataSize[4] = 640000;

    setOfR[0] = 0.002;
    setOfR[1] = 0.004;
    setOfR[2] = 0.006;
    setOfR[3] = 0.008;
    setOfR[4] = 0.01;

    setOfMinPts[0] = 4;
    setOfMinPts[1] = 8;
    setOfMinPts[2] = 16;
    setOfMinPts[3] = 32;
    setOfMinPts[4] = 64;

    defaultMin = 8;
    defaultR = 0.008;
    defaultPts = 160000;

    defaultP = 30;
    datasetPath = "/data/dbscan/Porto_taxi_data.csv";
  }

  if (NGSI) {
    setOfDataSize[0] = 50000;
    setOfDataSize[1] = 100000;
    setOfDataSize[2] = 200000;
    setOfDataSize[3] = 400000;
    setOfDataSize[4] = 800000;

    setOfR[0] = 0.5;
		setOfR[1] = 0.75;
		setOfR[2] = 1;
		setOfR[3] = 1.25;
		setOfR[4] = 1.5;

    setOfMinPts[0] = 4;
    setOfMinPts[1] = 8;
    setOfMinPts[2] = 16;
    setOfMinPts[3] = 32;
    setOfMinPts[4] = 64;

    defaultMin = 8;
    defaultR = 1.25;
    defaultPts = 400000;

    defaultP = 100;
    datasetPath = "/home/mpoudel/datasets/NGSIM_Data.txt";
  }

  if (SPATIAL) {
    setOfDataSize[0] = 25000;
    setOfDataSize[1] = 50000;
    setOfDataSize[2] = 100000;
    setOfDataSize[3] = 200000;
    setOfDataSize[4] = 400000;

    setOfR[0] = 0.002;
    setOfR[1] = 0.004;
    setOfR[2] = 0.006;
    setOfR[3] = 0.008;
    setOfR[4] = 0.01;

    setOfMinPts[0] = 4;
    setOfMinPts[1] = 8;
    setOfMinPts[2] = 16;
    setOfMinPts[3] = 32;
    setOfMinPts[4] = 64;

    defaultMin = 8;
    defaultR = 0.008;
    defaultPts = 400000;

    defaultP = 80;
    datasetPath = "/home/mpoudel/datasets/3D_spatial_network.csv";
  }

  if (IONO2D) {
    setOfDataSize[0] = 50000;
    setOfDataSize[1] = 100000;
    setOfDataSize[2] = 200000;
    setOfDataSize[3] = 400000;
    setOfDataSize[4] = 800000;

    setOfR[0] = 0.5;
    setOfR[1] = 0.75;
    setOfR[2] = 1;
    setOfR[3] = 1.25;
    setOfR[4] = 1.5;

    setOfMinPts[0] = 4;
    setOfMinPts[1] = 8;
    setOfMinPts[2] = 16;
    setOfMinPts[3] = 32;
    setOfMinPts[4] = 64;

    defaultMin = 4;
    defaultR = 1.5;
    defaultPts = 400000;

    defaultP = 80;
    datasetPath = "/data/geodata/iono_20min_2Mpts_2D.txt";
  }

  if (SPATIAL3D) {
    setOfDataSize[0] = 25000;
    setOfDataSize[1] = 50000;
    setOfDataSize[2] = 100000;
    setOfDataSize[3] = 200000;
    setOfDataSize[4] = 400000;

    setOfR[0] = 0.02;
    setOfR[1] = 0.04;
    setOfR[2] = 0.06;
    setOfR[3] = 0.08;
    setOfR[4] = 0.1;

    setOfMinPts[0] = 1;
    setOfMinPts[1] = 2;
    setOfMinPts[2] = 4;
    setOfMinPts[3] = 8;
    setOfMinPts[4] = 16;

    defaultMin = 2;
    defaultR = 0.08;
    defaultPts = 400000;

    defaultP = 10;
    datasetPath = "/home/mpoudel/datasets/3D_spatial_network.txt";
  }

  if (IONO3D) {
    setOfDataSize[0] = 100000;
    setOfDataSize[1] = 200000;
    setOfDataSize[2] = 400000;
    setOfDataSize[3] = 800000;
    setOfDataSize[4] = 1600000;

    setOfR[0] = 0.5;
    setOfR[1] = 0.75;
    setOfR[2] = 1;
    setOfR[3] = 1.25;
    setOfR[4] = 1.5;

    setOfMinPts[0] = 4;
    setOfMinPts[1] = 8;
    setOfMinPts[2] = 16;
    setOfMinPts[3] = 32;
    setOfMinPts[4] = 64;

    defaultMin = 4;
    defaultR = 1.5;
    defaultPts = 400000;

    defaultP = 40;
    datasetPath = "/data/geodata/iono_20min_2Mpts_3D.txt";
  }


    // // Different set of Eps
    // printf("################ EPS IMPACT ################\n");
    // for (int i = 0; i < 5; i++) {
    //   runDBSCAN(datasetPath, defaultPts, setOfR[i], defaultMin, defaultP);
    // }

    // // Different set of MinPts
    // printf("################ MINPTS IMPACT ################\n");
    // for (int i = 0; i < 5; i++) {
    //   runDBSCAN(datasetPath, defaultPts, defaultR, setOfMinPts[i], defaultP);
    // }
  
    // Different set of Points
    printf("################ POINTS IMPACT ################\n");
    for (int i = 0; i < 5; i++) {
      runDBSCAN(datasetPath, setOfDataSize[i], defaultR, defaultMin, defaultP);
    }


    // setOfP[0] = 10;
    // setOfP[1] = 20;
    // setOfP[2] = 30;
    // setOfP[3] = 40;
    // setOfP[4] = 50;
    
    // printf("################ PARTITION - POINTS IMPACT ################\n");
    // for (int i = 0; i < 5; i++) {
    //   for (int j = 0; j < 5; j++) {
    //     runDBSCAN(datasetPath, setOfDataSize[i], defaultR, defaultMin, setOfP[j]);
    //   }
    // }

    // printf("################ PARTITION - EPS IMPACT ################\n");
    // for (int i = 0; i < 5; i++) {
    //   for (int j = 0; j < 5; j++) {
    //     runDBSCAN(datasetPath, defaultPts, setOfR[i], defaultMin, setOfP[j]);
    //   }
    // }

  
}
